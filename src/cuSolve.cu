#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hipsolver.h>
#include <hipsparse.h>

#include <cstddef>
#include <iostream>
#include <tuple>
#include <unordered_map>
#include <vector>

#include "Mesh.h"
#include "Timer.h"

// CUDA API error checking
#define CUDA_CHECK(err)                                                   \
    do {                                                                  \
        hipError_t err_ = (err);                                         \
        if (err_ != hipSuccess) {                                        \
            printf("CUDA error %d at %s:%d\n", err_, __FILE__, __LINE__); \
            throw std::runtime_error("CUDA error");                       \
        }                                                                 \
    } while (0)

// cusolver API error checking
#define CUSOLVER_CHECK(err)                                                   \
    do {                                                                      \
        hipsolverStatus_t err_ = (err);                                        \
        if (err_ != HIPSOLVER_STATUS_SUCCESS) {                                \
            printf("cusolver error %d at %s:%d\n", err_, __FILE__, __LINE__); \
            throw std::runtime_error("cusolver error");                       \
        }                                                                     \
    } while (0)

// cusparse API error checking
#define CUSPARSE_CHECK(err)                                                   \
    do {                                                                      \
        hipsparseStatus_t err_ = (err);                                        \
        if (err_ != HIPSPARSE_STATUS_SUCCESS) {                                \
            printf("cusparse error %d at %s:%d\n", err_, __FILE__, __LINE__); \
            throw std::runtime_error("cusparse error");                       \
        }                                                                     \
    } while (0)

std::unordered_map<Key, double, KeyHash, KeyEqual>
Mesh::getStiffnessMatrixMap() {
    std::unordered_map<Key, double, KeyHash, KeyEqual> stiffnessMatrixMap;
    for (auto&& element : Elements) {
        auto&& ke = element->getStiffnessMatrix();
        for (size_t i = 0; i < 3; ++i) {
            for (size_t j = 0; j < 3; ++j) {
                size_t l = element->nodes[i]->getIndex() * 2;
                size_t m = element->nodes[j]->getIndex() * 2;
                stiffnessMatrixMap[Key(l, m)] += ke(2 * i, 2 * j);
                stiffnessMatrixMap[Key(l + 1, m + 1)] +=
                    ke(2 * i + 1, 2 * j + 1);
            }
        }
    }
    return stiffnessMatrixMap;
}

int Mesh::cuSolver(std::list<Load>& loads, std::list<Boundary>& boundaries,
                   bool verbose) {
    Timer timer;

    int numRows = Nodes.size() * 2;

    // Get equivalent force
    Force.resize(Nodes.size() * 2);
    Force.setZero();
    double* hostForce = (double*)(malloc(sizeof(double) * numRows));
    for (auto&& load : loads) {
        auto&& equivalentForce = Mesh::equivalentForce(&load);
        for (size_t i = 0; i < 3; ++i) {
            size_t j = 2 * load.nodes[i]->getIndex();
            Force.coeffRef(j) += equivalentForce[2 * i];
            Force.coeffRef(j + 1) += equivalentForce[2 * i + 1];
            hostForce[j] += equivalentForce[2 * i];
            hostForce[j + 1] += equivalentForce[2 * i + 1];
        }
    }
    if (verbose) {
        std::cout << "  Equivalent force calculated in " << timer << std::endl;
        timer.reset();
    }

    // Assemble stiffness matrix
    auto&& kMap = getStiffnessMatrixMap();
    int nnz = kMap.size();
    if (verbose) {
        std::cout << "  Stiffness matrix assembled in " << timer << std::endl;
        timer.reset();
    }

    // Apply boundary conditions
    {
        for (auto&& boundary : boundaries) {
            for (size_t i = 0; i < 2; ++i) {
                if (boundary.fixed[i]) {
                    size_t j = 2 * boundary.node->getIndex() + i;
                    // kMap[Key(j,j)] = std::numeric_limits<double>::max();
                    kMap[Key(j, j)] *= 1e50;
                    Force.coeffRef(j) = 0;
                    hostForce[j] = 0;
                }
            }
        }
    }

    std::vector<int> hostCsrRowPtr;
    std::vector<int> hostCsrColInd;
    std::vector<double> hostCsrValues;
    int currentRow = 0;
    hostCsrRowPtr.push_back(0);
    for (auto& elem : kMap) {
        int row, col;
        double val;
        std::tie(row, col) = elem.first;
        val = elem.second;

        // Row pointer update
        while (currentRow <= row) {
            hostCsrRowPtr.push_back(hostCsrColInd.size());
            currentRow++;
        }

        // Fill column index and value arrays
        hostCsrColInd.push_back(col);
        hostCsrValues.push_back(val);
    }
    // Deal with the last row
    while (currentRow < numRows) {
        hostCsrRowPtr.push_back(hostCsrColInd.size());
        currentRow++;
    }

    // cuSolver
    hipsolverSpHandle_t cusolverSpHandle;
    csrqrInfo_t info = NULL;
    hipsparseMatDescr_t descrA = NULL;
    hipStream_t stream = NULL;
    CUSOLVER_CHECK(hipsolverSpCreate(&cusolverSpHandle));
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUSOLVER_CHECK(hipsolverSpSetStream(cusolverSpHandle, stream));

    CUSPARSE_CHECK(hipsparseCreateMatDescr(&descrA));

    CUSPARSE_CHECK(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
    CUSPARSE_CHECK(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE));

    CUSOLVER_CHECK(cusolverSpCreateCsrqrInfo(&info));

    // Allocate device memory
    double* deviceForce;
    int* deviceCsrRowPtr;
    int* deviceCsrColInd;
    double* deviceCsrValues;
    double* deviceDisplacement;

    CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&deviceCsrValues),
                          sizeof(double) * nnz));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&deviceCsrRowPtr),
                          sizeof(int) * (numRows + 1)));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&deviceCsrColInd),
                          sizeof(int) * nnz));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&deviceForce),
                          sizeof(double) * numRows));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&deviceDisplacement),
                          sizeof(double) * numRows));

    int* singularity;
    CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&singularity), sizeof(int)));

    CUDA_CHECK(hipMemcpy(deviceCsrValues, hostCsrValues.data(),
                          sizeof(double) * nnz, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(deviceCsrRowPtr, hostCsrRowPtr.data(),
                            sizeof(int) * (numRows + 1), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(deviceCsrColInd, hostCsrColInd.data(),
                            sizeof(int) * nnz, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(deviceForce, hostForce, sizeof(double) * numRows,
                            hipMemcpyHostToDevice));

    CUSOLVER_CHECK(cusolverSpDcsrlsvluHost(
        cusolverSpHandle, numRows, nnz, descrA, deviceCsrValues,
        deviceCsrRowPtr, deviceCsrColInd, deviceForce, 1e-10, 0,
        deviceDisplacement, singularity));

    double* hostDisplacement = (double*)(malloc(sizeof(double) * numRows));
    CUDA_CHECK(hipMemcpy(hostDisplacement, deviceDisplacement, sizeof(double) * numRows,
                          hipMemcpyDeviceToHost));
    
    hipFree(deviceCsrValues);
    hipFree(deviceCsrRowPtr);
    hipFree(deviceCsrColInd);
    hipFree(deviceForce);
    hipFree(deviceDisplacement);
    hipFree(singularity);
    hipsolverSpDestroy(cusolverSpHandle);

    
    // Copy displacement to Nodes
    for (size_t i = 0; i < Nodes.size(); ++i) {
        Nodes[i]->Displacement(0) = hostDisplacement[2 * i];
        Nodes[i]->Displacement(1) = hostDisplacement[2 * i + 1];
    }

    // Calculate stain stress
    for (auto&& element : Elements) {
        element->calculateStrainStressGaussPoint();
    }

    return 0;
}  